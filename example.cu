#include "hip/hip_runtime.h"
#include "thrust/device_ptr.h"
#include "thrust/sort.h"

__global__ void calculate_hash(uint *hash_values, uint *particle_ids, int length)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if(i >= length)
        return;

    hash_values[i] =  1;
    particle_ids[i] = i;
}

void hash_particles_gpu(uint *d_hash_values, uint *d_particle_ids, int length)
{
    int block_size = 256;
    int num_blocks = ceil(length/(float)block_size);

    calculate_hash<<<num_blocks, block_size>>>(d_hash_values, d_particle_ids, length);  

    hipDeviceSynchronize();

    thrust::device_ptr<uint> keys(d_hash_values);
    thrust::device_ptr<uint> values(d_particle_ids);
    thrust::sort_by_key(keys, keys+length, values);
}

int main(int argc, char *argv[])
{
    int length = 15;
    int bytes;

    #ifdef BROKE
    int *m_int;
    hipMallocManaged((void**)&m_int, sizeof(int));
    #endif

    // Allocate uint hash value array
    bytes = length*sizeof(unsigned int);
    unsigned int * hash_values;
    hipMalloc((void**)&hash_values, bytes);    

    // Allocate uint particle ID array
    bytes = length*sizeof(unsigned int);
    unsigned int *particle_ids;
    hipMalloc((void**)&particle_ids, bytes);

    hash_particles_gpu(hash_values, particle_ids, length);
}
